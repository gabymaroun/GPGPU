#include <stdio.h>
#include <hip/hip_runtime.h>


// Initialize host vectors
void init(int *a, int *b, int n) {
  for (int i=0; i < n; ++i) {
    a[i] = i;
    b[i] = n-i;
  }
}

// Check result correctness
void check(int *c, int n) {
  int i = 0;
  while (i < n && c[i] == n) {
    ++i;
  }
  if (i == n)
    printf("Ok\n");
  else
    printf("Non ok\n");
}


// Cuda kernel
__global__ void add(int *a, int *b, int *c, int n) {
  //@TODO@ : complete kernel code
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if(i < n) {
		c[i]=a[i]+b[i];
	}
}

int main(int argc, char **argv)
{	

  if(argc<2) {printf("Give the vector size as first parameter\n");;exit(2);}


  int n = atoi(argv[1]);
  int b = atoi(argv[2]);
  int t = atoi(argv[3]);

  printf("Vector size is %d\n",n);
  // host pointers
  int *host_a, *host_b, *host_c;
  // Device pointers
  int *dev_a, *dev_b, *dev_c;

  // Allocations on host
  //@TODO@ : 
	host_a= (int *) malloc(sizeof(int)*n);
	host_b= (int *) malloc(sizeof(int)*n);
	host_c= (int *) malloc(sizeof(int)*n);

  // Initialize vectors
  init(host_a,host_b,n);

  // Allocations on device
  //@TODO@ : 
	hipMalloc(&dev_a, sizeof(int) *n);
	hipMalloc(&dev_b, sizeof(int) *n);
	hipMalloc(&dev_c, sizeof(int) *n);


  // Copy from host to device
  //@TODO@ : complete here
	hipMemcpy(dev_a, host_a, sizeof(int)*n,hipMemcpyHostToDevice);
	hipMemcpy(dev_b, host_b, sizeof(int)*n,hipMemcpyHostToDevice);

  // Invoke kernel
  //@TODO@ : complete here
	add<<<b, t>>>(dev_a, dev_b, dev_c, n );

  // Copy result from device to host
  //@TODO@ : complete here
	hipMemcpy(host_c, dev_c, sizeof(int)*n, hipMemcpyDeviceToHost);

  // Check result
  check(host_c,n);

  // Free device memory
  hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);
  // Free host memory
  free(host_a); free(host_b); free(host_c);
  return 0;
}
