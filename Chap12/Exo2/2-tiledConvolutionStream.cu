#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <string>
//#include <time.h>

#include "img_utils.hpp"
/*typedef unsigned long long ttype;
ttype gettime(void) {
  struct timespec ts;
  clock_gettime(CLOCK_MONOTONIC, &ts);
  return (((ttype)ts.tv_sec*1e9) + ts.tv_nsec);
}*/


//#define maskCols 5
//#define maskRows 5
//#define filter_width  5
// #define Mask_radius Mask_width/2
#define TILE_WIDTH 32
// #define w (TILE_WIDTH + filter_width - 1)
#define clamp(x) (min(max((x), 0.0), 1.0))

using namespace std;

//@TODO@ : Write the kernel here

__global__ void convolution_2D_tiled_kernel(float *I, const float* __restrict__ M, float *P, int channels, int width, int height, int filter_width) {

   

   int k;
   int Mask_radius = filter_width/2;
   int w= (TILE_WIDTH + filter_width - 1);
   
   extern __shared__ float N_ds[];
   
   for (k = 0; k < channels; k++) {
      // First batch loading
      int dest = threadIdx.y * TILE_WIDTH + threadIdx.x,
         destY = dest / w,
         destX = dest % w,
         srcY = blockIdx.y * TILE_WIDTH + destY - Mask_radius,
         srcX = blockIdx.x * TILE_WIDTH + destX - Mask_radius,
         src = (srcY * width + srcX) * channels + k;
         
      if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
         N_ds[destY*w+destX] = I[src];
      else
         N_ds[destY*w+destX] = 0;

      // Second batch loading
      int dest1 = dest + TILE_WIDTH * TILE_WIDTH;
      destY = dest1 / w, 
      destX = dest1 % w,
      srcY = blockIdx.y * TILE_WIDTH + destY - Mask_radius,
      srcX = blockIdx.x * TILE_WIDTH + destX - Mask_radius,
      src = (srcY * width + srcX) * channels + k;
      
      if (destY < w) {
         if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
            N_ds[destY*w+destX] = I[src];
         else
            N_ds[destY*w+destX] = 0;
      }
      __syncthreads();

      float accum = 0;
      int y, x;
      for (y = 0; y < filter_width; y++)
         for (x = 0; x < filter_width; x++)
            accum += N_ds[(threadIdx.y + y)*w+(threadIdx.x + x)] * M[y * filter_width + x];
      y = blockIdx.y * TILE_WIDTH + threadIdx.y;
      x = blockIdx.x * TILE_WIDTH + threadIdx.x;
      if (y < height && x < width)
         P[(y * width + x) * channels + k] = accum;
      __syncthreads();
   }
}

int main(int argc, char **argv)
{
	if(argc!=3) {
	cout<<"Program takes two image filenames as parameters"<<endl;
	exit(3);
	}

	float *imgIn, *imgOut;
	int nCols, nRows, channels;
	
	
	// Allocate images and initialize from file
	imgIn = read_image_asfloat(argv[1],&nCols, &nRows, &channels);
	int imgSize = nCols * nRows * channels;
	


	//imgOut = (float *)calloc(nCols * nRows * channels, sizeof(float));	


  // Allocations on host
  //@TODO@ : 
	hipHostAlloc((void **) &imgOut, imgSize * sizeof(int), hipHostMallocDefault);

                                        
	// blur mask
   int filter_width = 7; 
   float maskData[filter_width * filter_width];
   for(int i=0; i<filter_width * filter_width; i++) 
     maskData[i] = 1.0/((float)filter_width * filter_width); 

  
     
	// Allocates device images
	//float *d_imgIn, *d_imgOut, *d_MaskData;
	const int blockSize = 256, nStreams = 4;
   // const int n = imgSize * blockSize * nStreams;
   int n = blockSize * nStreams;
   const int streamSize = imgSize * blockSize * nStreams;
/*   const int streamBytes = streamSize * sizeof(float);*/
   int img_out_start;
	int img_out_end;
	int img_in_start;
	int img_in_end;
	int Mask_radius = filter_width/2;

   //const int bytes = n * sizeof(float);
   
	hipStream_t streams[nStreams];
	float *d_imgIn[nStreams];
	float *d_imgOut[nStreams];
	float *d_MaskData;


	hipMalloc(&d_MaskData, filter_width * filter_width * sizeof(float));
	
	hipMemcpy(d_MaskData, maskData, filter_width * filter_width * sizeof(float), hipMemcpyHostToDevice );
	
	 for(int i=0;i<nStreams;i++)
  {
  		
		hipStreamCreate(&streams[i]); 
		//@TODO@ : Complete for device allocations
		hipMalloc(&d_imgIn[i], nCols * nRows * channels * sizeof(float));
		hipMalloc(&d_imgOut[i], nCols * nRows * channels * sizeof(float));

	}


	//int dim=32;
	dim3 DimGrid(1 + (nCols-1)/TILE_WIDTH, 1 + (nRows-1)/TILE_WIDTH);
	//int w = TILE_WIDTH - filter_width + 1;
	dim3 DimBlock(TILE_WIDTH, TILE_WIDTH);
	int size = (TILE_WIDTH * TILE_WIDTH)*channels*sizeof (float);
	/*dim3 DimGrid((nCols-1)/dim+ 1, (nRows-1)/dim+1, 1);
	dim3 DimBlock(dim, dim, 1);*/
	/*size_t nbytes = DimBlock.x*DimBlock.y*sizeof(float);*/
	
	for (int i=0; i<n; i+= streamSize) 
	{
	  for(int j=0;j<nStreams;j++)
	 {
		
		if (img_out_start<nRows){
			
			if(img_out_start+n <= nRows)
				img_out_end=img_out_start+n;
			else img_out_end=nRows;

			 
			if(img_out_start - Mask_radius >= 0)
				img_in_start = img_out_start - Mask_radius;
			else img_in_start = 0;
	
			if(img_out_end + Mask_radius <= nRows)
				img_in_end = img_out_end + Mask_radius;
			else img_in_end=nRows;

			hipMemcpyAsync(d_imgIn[j], imgIn +img_in_start*nCols*channels , (img_in_end - img_in_start)*channels*nCols*sizeof(float), hipMemcpyHostToDevice, streams[j]);

			convolution_2D_tiled_kernel<<<DimGrid,DimBlock,size,streams[j]>>>( d_imgIn[j], d_MaskData, d_imgOut[j],channels, nCols, nRows, filter_width);
			
			hipMemcpyAsync(imgOut+img_out_start*nCols*channels, d_imgOut[j] , (img_out_end - img_out_start)*channels*nCols*sizeof(float), hipMemcpyDeviceToHost, streams[j]);
			
			}
			img_out_start=img_out_start+n+1;

		}

	}
	hipDeviceSynchronize();

	// Write gray image to file
	write_image_fromfloat(argv[2], imgOut, nCols, nRows, channels);

	// Free device memory
  for(int i=0;i<nStreams;i++)
	{
		hipStreamDestroy(streams[i]);
		hipFree(d_imgIn[i]); 
		hipFree(d_imgOut[i]); 

	}
	hipFree(d_MaskData);

	
	// Free host memory
	hipHostFree(imgIn); 
	hipHostFree(imgOut); 

	
	return 0;
}
